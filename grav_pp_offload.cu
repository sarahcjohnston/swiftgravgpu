#include "hip/hip_runtime.h"
#include "externalfunctions.cu"
#include "multipole_struct.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

/* Local Cuda includes */
#include "src/cuda_streams.h"
#include "src/gpu_params.h"

extern "C" void gpu_device_props(struct gpu_info *gpu_info) {

  /* Set the device ID */
  hipGetDevice(&gpu_info->device_id);

  /* Get the device properties */
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, gpu_info->device_id);

  /* Set the number of streaming multiprocessors */
  gpu_info->nr_sm = deviceProp.multiProcessorCount;

  /* Set the maximum number of threads per SM */
  gpu_info->max_threads_per_sm = deviceProp.maxThreadsPerMultiProcessor;

  /* Set the maximum number of threads per block */
  gpu_info->max_threads_per_block = deviceProp.maxThreadsPerBlock;

  /* Set the maximum number of blocks per SM */
  gpu_info->max_blocks_per_sm = deviceProp.maxBlocksPerMultiProcessor;

  /* Set the maximum amount of shared memory per SM */
  gpu_info->max_shared_memory_per_sm = deviceProp.sharedMemPerMultiprocessor;

  /* Set the maximum amount of shared memory per block */
  gpu_info->max_shared_memory_per_block = deviceProp.sharedMemPerBlock;

  /* Set the maximum number of registers per block */
  gpu_info->max_registers_per_block = deviceProp.regsPerBlock;

  /* Set the warp size */
  gpu_info->warp_size = deviceProp.warpSize;

  /* Set the maximum number of threads per block dimension */
  gpu_info->max_threads_per_block_dimension = deviceProp.maxThreadsDim[0];

  /* Set the maximum grid size */
  gpu_info->max_grid_size = deviceProp.maxGridSize[0];

  /* Set the maximum number of threads per block dimension x */
  gpu_info->max_threads_per_block_dimension_x = deviceProp.maxThreadsDim[0];

  /* Set the maximum number of threads per block dimension y */
  gpu_info->max_threads_per_block_dimension_y = deviceProp.maxThreadsDim[1];

  /* Set the maximum number of threads per block dimension z */
  gpu_info->max_threads_per_block_dimension_z = deviceProp.maxThreadsDim[2];
}

// PP ALL INTERACTIONS
__global__ void pair_grav_pp(
    int periodic, const float *CoM_i, const float *CoM_j, float rmax_i,
    float rmax_j, double min_trunc, int *active_i, int *mpole_i, int *active_j,
    int *mpole_j, float dim_0, float dim_1, float dim_2, float *h_i, float *h_j,
    float *mass_i_arr, float *mass_j_arr, const float r_s_inv, const float *x_i,
    const float *x_j, const float *y_i, const float *y_j, const float *z_i,
    const float *z_j, float *a_x_i, float *a_y_i, float *a_z_i, float *a_x_j,
    float *a_y_j, float *a_z_j, float *pot_i, float *pot_j, int gcount_i,
    int gcount_padded_i, int gcount_j, int gcount_padded_j, int ci_active,
    int cj_active, const int symmetric, const int allow_mpole,
    const struct multipole *restrict multi_i,
    const struct multipole *restrict multi_j, float *epsilon,
    const int allow_multipole_j, const int allow_multipole_i) {

  int max_r_decision = 0;

  /* Can we use the Newtonian version or do we need the truncated one ? */

  /* Not periodic -> Can always use Newtonian potential */
  /* Let's updated the active cell(s) only */

  /* First the P2P */
  grav_pp_full(active_i, mpole_i, dim_0, dim_1, dim_2, h_i, h_j, mass_j_arr,
               r_s_inv, x_i, x_j, y_i, y_j, z_i, z_j, a_x_i, a_y_i, a_z_i,
               pot_i, gcount_i, gcount_padded_j, periodic, ci_active, 0,
               symmetric, max_r_decision);

  /* Then the M2P */
  grav_pm_full(active_i, mpole_i, gcount_padded_i, CoM_j, multi_j, periodic,
               dim_0, dim_1, dim_2, x_i, y_i, z_i, gcount_i, a_x_i, a_y_i,
               a_z_i, *h_i, pot_i, allow_multipole_j, allow_multipole_i,
               ci_active, 0, symmetric, max_r_decision);

  /* First the P2P */
  grav_pp_full(active_j, mpole_j, dim_0, dim_1, dim_2, h_j, h_i, mass_i_arr,
               r_s_inv, x_j, x_i, y_j, y_i, z_j, z_i, a_x_j, a_y_j, a_z_j,
               pot_j, gcount_j, gcount_padded_i, periodic, 0, cj_active,
               symmetric, max_r_decision);

  /* Then the M2P */
  grav_pm_full(active_j, mpole_j, gcount_padded_j, CoM_i, multi_i, periodic,
               dim_0, dim_1, dim_2, x_j, y_j, z_j, gcount_j, a_x_j, a_y_j,
               a_z_j, *h_j, pot_j, allow_multipole_i, allow_multipole_j, 0,
               cj_active, symmetric, max_r_decision);

  /* Periodic BC */

  /* Get the relative distance between the CoMs */
  double d[3] = {CoM_j[0] - CoM_i[0], CoM_j[1] - CoM_i[1], CoM_j[2] - CoM_i[2]};

  /* Correct for periodic BCs */
  d[0] = nearestf1(d[0], dim_0);
  d[1] = nearestf1(d[1], dim_1);
  d[2] = nearestf1(d[2], dim_2);

  const double r2 = d[0] * d[0] + d[1] * d[1] + d[2] * d[2];

  /* Get the maximal distance between any two particles */
  const double max_r = sqrt(r2) + rmax_i + rmax_j;

  if (max_r > min_trunc) {
    max_r_decision = 0;
  } else {
    max_r_decision = 1;
  }

  /* Do we need to use the truncated interactions ? */

  /* Periodic but far-away cells must use the truncated potential */

  /* Let's updated the active cell(s) only */

  /* First the (truncated) P2P */
  grav_pp_truncated(active_i, mpole_i, dim_0, dim_1, dim_2, h_i, h_j,
                    mass_j_arr, r_s_inv, x_i, x_j, y_i, y_j, z_i, z_j, a_x_i,
                    a_y_i, a_z_i, pot_i, gcount_i, gcount_padded_j, periodic,
                    ci_active, 0, symmetric, max_r_decision);

  /* Then the M2P */
  grav_pm_truncated(active_i, mpole_i, gcount_padded_i, CoM_j, multi_j,
                    periodic, dim_0, dim_1, dim_2, r_s_inv, x_i, y_i, z_i,
                    gcount_i, a_x_i, a_y_i, a_z_i, *h_i, pot_i,
                    allow_multipole_j, allow_multipole_i, ci_active, 0,
                    symmetric, max_r_decision);

  /* First the (truncated) P2P */
  grav_pp_truncated(active_j, mpole_j, dim_0, dim_1, dim_2, h_j, h_i,
                    mass_i_arr, r_s_inv, x_j, x_i, y_j, y_i, z_j, z_i, a_x_j,
                    a_y_j, a_z_j, pot_j, gcount_j, gcount_padded_i, periodic, 0,
                    cj_active, symmetric, max_r_decision);

  /* Then the M2P */
  grav_pm_truncated(active_j, mpole_j, gcount_padded_j, CoM_i, multi_i,
                    periodic, dim_0, dim_1, dim_2, r_s_inv, x_i, y_i, z_i,
                    gcount_j, a_x_j, a_y_j, a_z_j, *h_j, pot_j,
                    allow_multipole_j, allow_multipole_i, 0, cj_active,
                    symmetric, max_r_decision);

  /* Periodic but close-by cells can use the full Newtonian potential */

  /* Let's updated the active cell(s) only */

  /* First the (Newtonian) P2P */
  grav_pp_full(active_i, mpole_i, dim_0, dim_1, dim_2, h_i, h_j, mass_j_arr,
               r_s_inv, x_i, x_j, y_i, y_j, z_i, z_j, a_x_i, a_y_i, a_z_i,
               pot_i, gcount_i, gcount_padded_j, periodic, ci_active, 0,
               symmetric, max_r_decision);

  /* Then the M2P */
  grav_pm_full(active_i, mpole_i, gcount_padded_i, CoM_j, multi_j, periodic,
               dim_0, dim_1, dim_2, x_i, y_i, z_i, gcount_i, a_x_i, a_y_i,
               a_z_i, *h_i, pot_i, allow_multipole_j, allow_multipole_i,
               ci_active, 0, symmetric, max_r_decision);

  /* First the (Newtonian) P2P */
  grav_pp_full(active_j, mpole_j, dim_0, dim_1, dim_2, h_j, h_i, mass_i_arr,
               r_s_inv, x_j, x_i, y_j, y_i, z_j, z_i, a_x_j, a_y_j, a_z_j,
               pot_j, gcount_j, gcount_padded_i, periodic, 0, cj_active,
               symmetric, max_r_decision);

  /* Then the M2P */
  grav_pm_full(active_j, mpole_j, gcount_padded_j, CoM_i, multi_i, periodic,
               dim_0, dim_1, dim_2, x_j, y_j, z_j, gcount_j, a_x_j, a_y_j,
               a_z_j, *h_j, pot_j, allow_multipole_i, allow_multipole_j, 0,
               cj_active, symmetric, max_r_decision);
}

// do not touch these variables you dumbass you need them to be pointers girly
extern "C" void pp_offload(
    int periodic, const float *CoM_i, const float *CoM_j, float rmax_i,
    float rmax_j, double min_trunc, int *active_i, int *mpole_i, int *active_j,
    int *mpole_j, float *dim, const float *x_i, const float *x_j_arr,
    const float *y_i, const float *y_j_arr, const float *z_i,
    const float *z_j_arr, float *pot_i, float *pot_j, float *a_x_i,
    float *a_y_i, float *a_z_i, float *a_x_j, float *a_y_j, float *a_z_j,
    float *mass_i_arr, float *mass_j_arr, const float *r_s_inv, float *h_i,
    float *h_j_arr, const int *gcount_i, const int *gcount_padded_i,
    const int *gcount_j, const int *gcount_padded_j, int ci_active,
    int cj_active, const int symmetric, const int allow_mpole,
    const struct multipole *restrict multi_i,
    const struct multipole *restrict multi_j, float *epsilon,
    const int *allow_multipole_j, const int *allow_multipole_i,
    struct gpu_info *gpu_info) {

  float a_x_i_new[*gcount_i];
  float a_y_i_new[*gcount_i];
  float a_z_i_new[*gcount_i];
  float pot_i_new[*gcount_i];

  float a_x_j_new[*gcount_j];
  float a_y_j_new[*gcount_j];
  float a_z_j_new[*gcount_j];
  float pot_j_new[*gcount_j];

  // create device pointers
  float *d_h_i;
  float *d_h_j;
  float *d_mass_i;
  float *d_mass_j;
  float *d_x_i;
  float *d_x_j;
  float *d_y_i;
  float *d_y_j;
  float *d_z_i;
  float *d_z_j;
  float *d_a_x_i;
  float *d_a_y_i;
  float *d_a_z_i;
  float *d_a_x_j;
  float *d_a_y_j;
  float *d_a_z_j;
  float *d_pot_i;
  float *d_pot_j;
  int *d_active_i;
  int *d_mpole_i;
  int *d_active_j;
  int *d_mpole_j;
  float *d_CoM_i;
  float *d_CoM_j;

  /* Get a stream to use (for testing these run 0-3 inclusive), we'll
   * randomly select one for now. */
  int rand_stream = rand() % gpu_info->nr_streams;
  hipStream_t stream = streams->streams[rand_stream];

  // hipDeviceSynchronize();

  // hipMalloc(&h_multi_j, 13*sizeof(float));
  // hipMemcpyAsync(h_multi_j, multi_j, 13*sizeof(float),
  // hipMemcpyHostToDevice);
  multipole *d_multi_j;
  hipMallocAsync(&d_multi_j, sizeof(multipole), stream);
  hipMemcpyAsync(d_multi_j, multi_j, sizeof(multipole), hipMemcpyHostToDevice,
                  stream);
  multipole *d_multi_i;
  hipMallocAsync(&d_multi_i, sizeof(multipole), stream);
  hipMemcpyAsync(d_multi_i, multi_i, sizeof(multipole), hipMemcpyHostToDevice,
                  stream);

  // allocate memory on device
  hipMallocAsync(&d_h_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_h_j, *gcount_padded_j * sizeof(float), stream);
  hipMallocAsync(&d_mass_i, *gcount_padded_i * sizeof(float), stream);
  hipMallocAsync(&d_mass_j, *gcount_padded_j * sizeof(float), stream);
  hipMallocAsync(&d_x_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_x_j, *gcount_padded_j * sizeof(float), stream);
  hipMallocAsync(&d_y_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_y_j, *gcount_padded_j * sizeof(float), stream);
  hipMallocAsync(&d_z_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_z_j, *gcount_padded_j * sizeof(float), stream);
  hipMallocAsync(&d_a_x_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_a_y_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_a_z_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_a_x_j, *gcount_j * sizeof(float), stream);
  hipMallocAsync(&d_a_y_j, *gcount_j * sizeof(float), stream);
  hipMallocAsync(&d_a_z_j, *gcount_j * sizeof(float), stream);
  hipMallocAsync(&d_pot_i, *gcount_i * sizeof(float), stream);
  hipMallocAsync(&d_pot_j, *gcount_j * sizeof(float), stream);
  hipMallocAsync(&d_active_i, *gcount_i * sizeof(int), stream);
  hipMallocAsync(&d_mpole_i, *gcount_i * sizeof(int), stream);
  hipMallocAsync(&d_active_j, *gcount_j * sizeof(int), stream);
  hipMallocAsync(&d_mpole_j, *gcount_j * sizeof(int), stream);
  hipMallocAsync(&d_CoM_i, 3 * sizeof(float), stream);
  hipMallocAsync(&d_CoM_j, 3 * sizeof(float), stream);

  // copy data to device
  hipMemcpyAsync(d_h_i, h_i, *gcount_i * sizeof(float), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(d_h_j, h_j_arr, *gcount_padded_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_mass_i, mass_i_arr, *gcount_padded_i * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_mass_j, mass_j_arr, *gcount_padded_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_x_i, x_i, *gcount_i * sizeof(float), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(d_x_j, x_j_arr, *gcount_padded_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_y_i, y_i, *gcount_i * sizeof(float), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(d_y_j, y_j_arr, *gcount_padded_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_z_i, z_i, *gcount_i * sizeof(float), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(d_z_j, z_j_arr, *gcount_padded_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_a_x_i, a_x_i, *gcount_i * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_a_y_i, a_y_i, *gcount_i * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_a_z_i, a_z_i, *gcount_i * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_a_x_j, a_x_j, *gcount_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_a_y_j, a_y_j, *gcount_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_a_z_j, a_z_j, *gcount_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_pot_i, pot_i, *gcount_i * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_pot_j, pot_j, *gcount_j * sizeof(float),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_active_i, active_i, *gcount_i * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_mpole_i, mpole_i, *gcount_i * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_active_j, active_j, *gcount_j * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_mpole_j, mpole_j, *gcount_j * sizeof(int),
                  hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(d_CoM_i, CoM_i, 3 * sizeof(float), hipMemcpyHostToDevice,
                  stream);
  hipMemcpyAsync(d_CoM_j, CoM_j, 3 * sizeof(float), hipMemcpyHostToDevice,
                  stream);

  // printf("%.16f %.16f\n", x_i[0], y_i[0]);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) printf("Error1: %s\n", hipGetErrorString(err));

  // hipDeviceSynchronize();

  // call kernel function
  pair_grav_pp<<<gpu_info->sms_multiple * gpu_info->nr_sm,
                 gpu_info->threads_per_block, 0, stream>>>(
      periodic, d_CoM_i, d_CoM_j, rmax_i, rmax_j, min_trunc, d_active_i,
      d_mpole_i, d_active_j, d_mpole_j, dim[0], dim[1], dim[2], d_h_i, d_h_j,
      d_mass_i, d_mass_j, *r_s_inv, d_x_i, d_x_j, d_y_i, d_y_j, d_z_i, d_z_j,
      d_a_x_i, d_a_y_i, d_a_z_i, d_a_x_j, d_a_y_j, d_a_z_j, d_pot_i, d_pot_j,
      *gcount_i, *gcount_padded_i, *gcount_j, *gcount_padded_j, ci_active,
      cj_active, symmetric, allow_mpole, d_multi_i, d_multi_j, epsilon,
      *allow_multipole_j, *allow_multipole_i);

  // hipDeviceSynchronize();

  hipError_t err2 = hipGetLastError();
  if (err2 != hipSuccess) printf("Error2: %s\n", hipGetErrorString(err2));

  // copy data from device
  hipMemcpyAsync(&a_x_i_new, d_a_x_i, *gcount_i * sizeof(float),
                  hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&a_y_i_new, d_a_y_i, *gcount_i * sizeof(float),
                  hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&a_z_i_new, d_a_z_i, *gcount_i * sizeof(float),
                  hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&pot_i_new, d_pot_i, *gcount_i * sizeof(float),
                  hipMemcpyDeviceToHost, stream);

  hipMemcpyAsync(&a_x_j_new, d_a_x_j, *gcount_j * sizeof(float),
                  hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&a_y_j_new, d_a_y_j, *gcount_j * sizeof(float),
                  hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&a_z_j_new, d_a_z_j, *gcount_j * sizeof(float),
                  hipMemcpyDeviceToHost, stream);
  hipMemcpyAsync(&pot_j_new, d_pot_j, *gcount_j * sizeof(float),
                  hipMemcpyDeviceToHost, stream);

  // printf("%.16f %.16f %.16f %.16f\n", a_x_i_new[0], a_y_i_new[0],
  // a_z_i_new[0], pot_i_new[0]);

  hipStreamSynchronize(stream);

  hipError_t err3 = hipGetLastError();
  if (err3 != hipSuccess) printf("Error3: %s\n", hipGetErrorString(err3));

  for (int i = 0; i < *gcount_i; i++) {
    a_x_i[i] += a_x_i_new[i];
    a_y_i[i] += a_y_i_new[i];
    a_z_i[i] += a_z_i_new[i];
    pot_i[i] += pot_i_new[i];
  }

  for (int i = 0; i < *gcount_j; i++) {
    a_x_j[i] += a_x_j_new[i];
    a_y_j[i] += a_y_j_new[i];
    a_z_j[i] += a_z_j_new[i];
    pot_j[i] += pot_j_new[i];
  }

  /*printf("gpu gcount_i: %i ", *gcount_i);
  for (int i = 0; i < *gcount_i; i++){
          printf("%.16f ", a_x_i[i]);}
  printf("\n");*/

  // printf("%f %f %f \n", a_x_i[0], a_y_i[0], a_z_i[0]);

  // printf("RESULT2: %f %f %f %f ", a_x_new[0], a_y_new[0], a_z_new[0],
  // pot_new[0]);

  // free memory
  hipFreeAsync(d_h_i, stream);
  hipFreeAsync(d_h_j, stream);
  hipFreeAsync(d_mass_i, stream);
  hipFreeAsync(d_mass_j, stream);
  hipFreeAsync(d_x_i, stream);
  hipFreeAsync(d_x_j, stream);
  hipFreeAsync(d_y_i, stream);
  hipFreeAsync(d_y_j, stream);
  hipFreeAsync(d_z_i, stream);
  hipFreeAsync(d_z_j, stream);
  hipFreeAsync(d_a_x_i, stream);
  hipFreeAsync(d_a_y_i, stream);
  hipFreeAsync(d_a_z_i, stream);
  hipFreeAsync(d_a_x_j, stream);
  hipFreeAsync(d_a_y_j, stream);
  hipFreeAsync(d_a_z_j, stream);
  hipFreeAsync(d_pot_i, stream);
  hipFreeAsync(d_pot_j, stream);
  hipFreeAsync(d_active_i, stream);
  hipFreeAsync(d_mpole_i, stream);
  hipFreeAsync(d_active_j, stream);
  hipFreeAsync(d_mpole_j, stream);
  hipFreeAsync(d_CoM_i, stream);
  hipFreeAsync(d_CoM_j, stream);
  hipFreeAsync(d_multi_j, stream);
  hipFreeAsync(d_multi_i, stream);

  hipError_t err4 = hipGetLastError();
  if (err4 != hipSuccess) printf("Error4: %s\n", hipGetErrorString(err4));
}
